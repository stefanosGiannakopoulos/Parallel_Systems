#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipError_t err;

    // Get the number of devices
    err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printf("Error getting device count: %s\n", hipGetErrorString(err));
        return -1;
    }

    printf("Number of CUDA devices: %d\n", deviceCount);

    // Loop through each device
    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;

        // Get device properties
        err = hipGetDeviceProperties(&deviceProp, device);
        if (err != hipSuccess) {
            printf("Error getting properties for device %d: %s\n", device, hipGetErrorString(err));
            continue;
        }

        printf("\nDevice %d: \"%s\"\n", device, deviceProp.name);
        printf("  Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("  Total global memory: %lu bytes\n", deviceProp.totalGlobalMem);
        printf("  Shared memory per block: %lu bytes\n", deviceProp.sharedMemPerBlock);
        printf("  Registers per block: %d\n", deviceProp.regsPerBlock);
        printf("  Warp size: %d\n", deviceProp.warpSize);
        printf("  Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
        printf("  Max threads dimensions: [%d, %d, %d]\n",
               deviceProp.maxThreadsDim[0],
               deviceProp.maxThreadsDim[1],
               deviceProp.maxThreadsDim[2]);
        printf("  Max grid size: [%d, %d, %d]\n",
               deviceProp.maxGridSize[0],
               deviceProp.maxGridSize[1],
               deviceProp.maxGridSize[2]);
        printf("  Clock rate: %.2f MHz\n", deviceProp.clockRate / 1000.0);
        printf("  Memory clock rate: %.2f MHz\n", deviceProp.memoryClockRate / 1000.0);
        printf("  Memory bus width: %d bits\n", deviceProp.memoryBusWidth);
        printf("  Multiprocessor count: %d\n", deviceProp.multiProcessorCount);
        printf("  L2 cache size: %d bytes\n", deviceProp.l2CacheSize);
        printf("  Concurrent kernels: %s\n", deviceProp.concurrentKernels ? "Yes" : "No");
        printf("  ECC enabled: %s\n", deviceProp.ECCEnabled ? "Yes" : "No");
    }

    return 0;
}
