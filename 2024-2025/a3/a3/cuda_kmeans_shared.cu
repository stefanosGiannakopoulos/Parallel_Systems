#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "alloc.h"
#include "error.h"

#ifdef __HIPCC__
inline void checkCuda(hipError_t e) {
    if (e != hipSuccess) {
        // hipGetErrorString() isn't always very helpful. Look up the error
        // number in the hipError_t enum in hip/driver_types.h in the CUDA includes
        // directory for a better explanation.
        error("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

inline void checkLastCudaError() {
    checkCuda(hipGetLastError());
}
#endif

__device__ int get_tid() {
  return blockDim.x*blockIdx.x+threadIdx.x;  /* TODO: copy me from naive version... */
}
  
/* square of Euclid distance between two multi-dimensional points using column-base format */
__host__ __device__ inline static
double euclid_dist_2_transpose(int numCoords,
                               int numObjs,
                               int numClusters,
                               double *objects,     // [numCoords][numObjs]
                               double *clusters,    // [numCoords][numClusters]
                               int objectId,
                               int clusterId) {
  int i;
  double ans = 0.0;

  /* TODO: Copy me from transpose version*/
  double to_square = 0.0;

    /* TODO: Calculate the euclid_dist of elem=objectId of objects from elem=clusterId from clusters, but for column-base format!!! */
     for(i=0; i<numCoords; i++) {
	  to_square = objects[numObjs*i+objectId]-clusters[numClusters*i+clusterId] ;   
	  ans += to_square*to_square;		    			    
          } 
          return (ans);
}

__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          double *objects,           //  [numCoords][numObjs]
                          double *deviceClusters,    //  [numCoords][numClusters]
                          int *deviceMembership,          //  [numObjs]
                          double *devdelta) {
  extern __shared__ double shmemClusters[];

  /* TODO: Copy deviceClusters to shmemClusters so they can be accessed faster.
    BEWARE: Make sure operations is complete before any thread continues... */
    int i,j ; 
    int local_tid = threadIdx.x;

    for(i=local_tid;i<numClusters;i+=blockDim.x){
	for(j=0;j<numCoords;j++) shmemClusters[j*numClusters + i] = deviceClusters[j*numClusters + i] ;  
    }  
    __syncthreads();

  /* Get the global ID of the thread. */
  int tid = get_tid();

  /* TODO: Maybe something is missing here... should all threads run this? */
  if (tid<numObjs) {
    int index, i;
    double dist, min_dist;

    /* find the cluster id that has min distance to object */
    index = 0;
    /* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/
    min_dist = euclid_dist_2_transpose(numCoords,numObjs,numClusters,objects,shmemClusters,tid,0);              

    for (i = 1; i < numClusters; i++) {
      /* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId using clusters in shmem*/
      dist = euclid_dist_2_transpose(numCoords,numObjs,numClusters,objects,shmemClusters,tid,i);

      /* no need square root */
      if (dist < min_dist) { /* find the min and its array index */
        min_dist = dist;
        index = i;
      }
    }

    if (deviceMembership[tid] != index) {
      /* TODO: Maybe something is missing here... is this write safe? */
       atomicAdd(devdelta, 1.0);
    }

    /* assign the deviceMembership to object objectId */
    deviceMembership[tid] = index;
  }
}

//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */
void kmeans_gpu(double *objects,      /* in: [numObjs][numCoords] */
                int numCoords,    /* no. features */
                int numObjs,      /* no. objects */
                int numClusters,  /* no. clusters */
                double threshold,    /* % objects change membership */
                long loop_threshold,   /* maximum number of iterations */
                int *membership,   /* out: [numObjs] */
                double *clusters,   /* out: [numClusters][numCoords] */
                int blockSize) {
  double timing = wtime(), timing_internal, timer_min = 1e42, timer_max = 0;
  double timing_gpu, timing_cpu, timing_transfers, transfers_time = 0.0, cpu_time = 0.0, gpu_time = 0.0;
  int loop_iterations = 0;
  int i, j, index, loop = 0;
  int *newClusterSize; /* [numClusters]: no. objects assigned in each
                                new cluster */
  double delta = 0, *dev_delta_ptr;          /* % of objects change their clusters */
  /* TODO: Copy me from transpose version*/
  double **dimObjects = (double**) calloc_2d(numCoords, numObjs, sizeof(double)); //calloc_2d(...) -> [numCoords][numObjs]
  double **dimClusters = (double**) calloc_2d(numCoords, numClusters, sizeof(double));  //calloc_2d(...) -> [numCoords][numClusters]
  double **newClusters = (double**) calloc_2d(numCoords, numClusters, sizeof(double));  //calloc_2d(...) -> [numCoords][numClusters]

  double *deviceObjects;
  double *deviceClusters;
  int *deviceMembership;

  printf("\n|-----------Shared GPU Kmeans------------|\n\n");

  /* TODO: Copy me from transpose version*/
    for (i=0; i<numObjs; i++) {
	for(j=0; j<numCoords; j++) {
	       dimObjects[j][i] = objects[i*numCoords+j];
		  }
		} 

  /* pick first numClusters elements of objects[] as initial cluster centers*/
  for (i = 0; i < numCoords; i++) {
    for (j = 0; j < numClusters; j++) {
      dimClusters[i][j] = dimObjects[i][j];
    }
  }

  /* initialize membership[] */
  for (i = 0; i < numObjs; i++) membership[i] = -1;

  /* need to initialize newClusterSize and newClusters[0] to all 0 */
  newClusterSize = (int *) calloc(numClusters, sizeof(int));
  assert(newClusterSize != NULL);

  timing = wtime() - timing;
  printf("t_alloc: %lf ms\n\n", 1000 * timing);
  timing = wtime();
  const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize) ? blockSize : numObjs;
  const unsigned int numClusterBlocks = (numObjs + numThreadsPerClusterBlock -1) / numThreadsPerClusterBlock; /* TODO: Calculate Grid size, e.g. number of blocks. */
  
  /*	Define the shared memory needed per block.
      - BEWARE: We can overrun our shared memory here if there are too many
      clusters or too many coordinates!
      - This can lead to occupancy problems or even inability to run.
      - Your exercise implementation is not requested to account for that (e.g. always assume deviceClusters fit in shmemClusters */
  const unsigned int clusterBlockSharedDataSize = numClusters*numCoords*sizeof(double);

  hipDeviceProp_t deviceProp;
  int deviceNum;
  hipGetDevice(&deviceNum);
  hipGetDeviceProperties(&deviceProp, deviceNum);

  if (clusterBlockSharedDataSize > deviceProp.sharedMemPerBlock) {
    error("Your CUDA hardware has insufficient block shared memory to hold all cluster centroids\n");
  }

  checkCuda(hipMalloc(&deviceObjects, numObjs * numCoords * sizeof(double)));
  checkCuda(hipMalloc(&deviceClusters, numClusters * numCoords * sizeof(double)));
  checkCuda(hipMalloc(&deviceMembership, numObjs * sizeof(int)));
  checkCuda(hipMalloc(&dev_delta_ptr, sizeof(double)));

  timing = wtime() - timing;
  printf("t_alloc_gpu: %lf ms\n\n", 1000 * timing);
  timing = wtime();

  checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
                       numObjs * numCoords * sizeof(double), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(deviceMembership, membership,
                       numObjs * sizeof(int), hipMemcpyHostToDevice));
  timing = wtime() - timing;
  printf("t_get_gpu: %lf ms\n\n", 1000 * timing);
  timing = wtime();

  do {
    timing_internal = wtime();

    /* GPU part: calculate new memberships */

    timing_transfers = wtime();
    /* TODO: Copy clusters to deviceClusters
    checkCuda(hipMemcpy(...)); */
    checkCuda(hipMemcpy(deviceClusters, dimClusters[0], numClusters*numCoords*sizeof(double),  hipMemcpyHostToDevice));

    transfers_time += wtime() - timing_transfers;

    checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double)));

    timing_gpu = wtime();
    //printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
    find_nearest_cluster
    <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, deviceClusters, deviceMembership, dev_delta_ptr);

    hipDeviceSynchronize();
    checkLastCudaError();
    gpu_time += wtime() - timing_gpu;
    //printf("Kernels complete for itter %d, updating data in CPU\n", loop);

    timing_transfers = wtime();
    /* TODO: Copy deviceMembership to membership
        checkCuda(hipMemcpy(...)); */
         checkCuda(hipMemcpy(membership, deviceMembership, numObjs*sizeof(int), hipMemcpyDeviceToHost));   

    /* TODO: Copy dev_delta_ptr to &delta
      checkCuda(hipMemcpy(...)); */
    checkCuda(hipMemcpy(&delta, dev_delta_ptr, sizeof(double), hipMemcpyDeviceToHost));
   
    transfers_time += wtime() - timing_transfers;
    

    /* CPU part: Update cluster centers*/

    timing_cpu = wtime();
    for (i = 0; i < numObjs; i++) {
      /* find the array index of nestest cluster center */
      index = membership[i];

      /* update new cluster centers : sum of objects located within */
      newClusterSize[index]++;
      for (j = 0; j < numCoords; j++)
        newClusters[j][index] += objects[i * numCoords + j];
    }

    /* average the sum and replace old cluster centers with newClusters */
    for (i = 0; i < numClusters; i++) {
      for (j = 0; j < numCoords; j++) {
        if (newClusterSize[i] > 0)
          dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
        newClusters[j][i] = 0.0;   /* set back to 0 */
      }
      newClusterSize[i] = 0;   /* set back to 0 */
    }

    delta /= numObjs;
    //printf("delta is %f - ", delta);
    loop++;
    //printf("completed loop %d\n", loop);
    cpu_time += wtime() - timing_cpu;

    timing_internal = wtime() - timing_internal;
    if (timing_internal < timer_min) timer_min = timing_internal;
    if (timing_internal > timer_max) timer_max = timing_internal;
  } while (delta > threshold && loop < loop_threshold);

  /*TODO: Update clusters using dimClusters. Be carefull of layout!!! clusters[numClusters][numCoords] vs dimClusters[numCoords][numClusters] */
   for (i=0; i<numClusters; i++) {
	  for (j=0; j<numCoords; j++) {
		clusters[i*numCoords + j] = dimClusters[j][i];
			               }
			          }

  timing = wtime() - timing;
  printf("nloops = %d  : total = %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\t"
         "-> t_cpu_avg = %lf ms\n\t-> t_gpu_avg = %lf ms\n\t-> t_transfers_avg = %lf ms\n\n|-------------------------------------------|\n",
         loop, 1000 * timing, 1000 * timing / loop, 1000 * timer_min, 1000 * timer_max,
         1000 * cpu_time / loop, 1000 * gpu_time / loop, 1000 * transfers_time / loop);

  char outfile_name[1024] = {0};
  sprintf(outfile_name, "Execution_logs/silver1-V100_Sz-%lu_Coo-%d_Cl-%d.csv",
          numObjs * numCoords * sizeof(double) / (1024 * 1024), numCoords, numClusters);
  FILE *fp = fopen(outfile_name, "a+");
  if (!fp) error("Filename %s did not open succesfully, no logging performed\n", outfile_name);
  fprintf(fp, "%s,%d,%lf,%lf,%lf\n", "Shmem", blockSize, timing / loop, timer_min, timer_max);
  fclose(fp);

  checkCuda(hipFree(deviceObjects));
  checkCuda(hipFree(deviceClusters));
  checkCuda(hipFree(deviceMembership));

  free(dimObjects[0]);
  free(dimObjects);
  free(dimClusters[0]);
  free(dimClusters);
  free(newClusters[0]);
  free(newClusters);
  free(newClusterSize);

  return;
}

